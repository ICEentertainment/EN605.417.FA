#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <chrono>

static const int N = 10000;

__global__ void add(int *a, int *b, int *c) {
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	while(tid<N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x*gridDim.x; // We multiply how many threads per block * how many blocks there are and add that index to tid. 
	}
}
void addHost(int *a, int *b, int *c) {
	for(int i =0; i<N; i++){
		c[i] = a[i] + b[i];
	}
}
void outputCardInfo();

int main(int argc,char* argv[]) {
	outputCardInfo();
	int blocks = 3;
	int threads = 64;
	if (argc == 2) {
		blocks = atoi(argv[1]);
		printf("Blocks changed to:%i\n", blocks);
	}
	else if (argc == 3) {
		blocks = atoi(argv[1]);
		threads = atoi(argv[2]);
		printf("Blocks changed to:%i\n", blocks);
		printf("Threads changed to:%i\n", threads);
	}
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	
	//Populate our arrays with numbers.
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i*i;
	}

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	auto start = std::chrono::high_resolution_clock::now();
	add<<<blocks,threads>>> (dev_a, dev_b, dev_c);
	auto stop = std::chrono::high_resolution_clock::now();
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	auto startHost = std::chrono::high_resolution_clock::now();
	addHost(a, b, c);
	auto stopHost = std::chrono::high_resolution_clock::now();
	std::cout <<std::endl<< " Time elapsed GPU = " << std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count() << "ns\n";
	std::cout << " Time elapsed Host = " << std::chrono::duration_cast<std::chrono::nanoseconds>(stopHost - startHost).count() << "ns\n";
	return 0;

}

void outputCardInfo() {
	hipDeviceProp_t Card;
	int count;
	hipGetDeviceCount(&count);
	std::cout << count << std::endl;
	hipGetDeviceProperties(&Card, 0);
	std::cout << Card.name << std::endl;;
	std::cout << "MaxThreads: " << Card.maxThreadsPerBlock << " Major: " << Card.major << " Minor: " << Card.minor << " clock Rate: " << Card.clockRate << std::endl;
	std::cout << "total Mem: " << Card.totalGlobalMem << " Total const Mem: " << Card.totalConstMem << " Texture alignment:" << Card.textureAlignment << std::endl;
	std::cout << "Multi processor count	:" << Card.multiProcessorCount <<" Max Threads" << Card.maxThreadsPerBlock;
	printf("\n\n");
}
